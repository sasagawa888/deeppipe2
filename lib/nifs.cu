#include "hip/hip_runtime.h"
#include "erl_nif.h"
#include "hipblas.h"
#include "stdio.h"
#include "time.h"


#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define IDX3C(c,i,j,in_h,in_w) ((c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define IDX4C(n,c,i,j,in_c,in_h,in_w) ((n)*((in_c)*(in_h)*(in_w)) + (c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define IDX5C(t,n,c,i,j,in_n,in_c,in_h,in_w) ((t)*((in_n)*(in_c)*(in_h)*(in_w)) + (n)*((in_c)*(in_h)*(in_w)) + (c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define BREAK return(enif_make_int(env, 0));
#define PI 3.14159265358979323846
#define SIGMOID(x)  (1 / (1+exp(-1*x)))
#define CHECK(call)                                   \
{                                                     \
    const hipError_t error = call;                   \
    if (error != hipSuccess)                         \
    {                                                 \
        return enif_make_int(env,10000+(int)error);   \
    }                                                 \
}
#define CUBLAS(call)                                  \
{                                                     \
    const hipblasStatus_t error = call;                  \
    if (error != HIPBLAS_STATUS_SUCCESS)               \
    {                                                 \
        return enif_make_int(env,11000+(int)error);   \
    }                                                 \
}

__global__ void pooling_kernel(float *a, float *b, float *c, int st_h, int st_w, int in_c, int in_h, int in_w)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,h1,w1,h2,w2,in_h2,in_w2,start_h1,end_h1,start_w1,end_w1,max_h,max_w;
    float max,fmax_h,fmax_w; 
    n1 = bid;
    c1 = tid;
    in_h2 = in_h / st_h;
    in_w2 = in_w / st_w;
    
    for(w2=0;w2<in_w2;w2++){
        for(h2=0;h2<in_h2;h2++){
            max = -999999999.0;
            start_h1 = st_h*h2;
            end_h1 = st_h*(h2+1);
            start_w1 = st_w*w2;
            end_w1 = st_w*(w2+1);
            for(h1=start_h1;h1<end_h1;h1++){
                for(w1=start_w1;w1<end_w1;w1++){
                    if(a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)] >= max){
                        max = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)];
                        max_h = h1;
                        max_w = w1;
                    }
                }
            }
            b[IDX4C(n1,c1,h2,w2,in_c,in_h2,in_w2)] = max;
            fmax_h = (float)max_h;
            fmax_w = (float)max_w;
            c[IDX4C(n1,c1,h2,w2,in_c,in_h2,in_w2)] = fmax_h * 1000.0 + fmax_w; 
        }
    }
    
}
  
  /*
  1st arg in_n of tensor
  2nd arg in_c of tensor
  3rd arg in_h of tensor
  4th arg in_w of tensor
  5th arg binary of tensor
  6th arg stride 

  return list [ts1,ts2]
  ts1 is result data for forward
  ts2 is result data dor backward. this is sparse matrix 
  e.g. 
  |0.1,0.2,0.3,0.4|
  |0.5,0.6,0.7,0.8|
  |0.9,1.0,1.1,1.2|
  |1.3,1.4,1.5,1.6|
  
  ts1
  |0.6,0.8|
  |1.4,1.6|

  ts2
  each element is  row*1000+col
  |1.0*1000+1.0,1.0*1000*3.0|
  |3.0*1000+1.0,3.0*1000+3.0|
  
  */
static ERL_NIF_TERM
pooling1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin,c_bin,tuple;
    int in_n,in_c,in_h,in_w,st_h,st_w, n1, n2;
    float *a,*b, *c;
    float *dev_a, *dev_b, *dev_c;
  
    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
    if (!enif_get_int(env, argv[5], &st_h)) return enif_make_int(env,6);
    if (!enif_get_int(env, argv[6], &st_w)) return enif_make_int(env,7);

    n1 = in_n * in_c * in_h * in_w;
    n2 = in_n * in_c * (in_h / st_h) * (in_w / st_w);
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env,  n2 * sizeof(float), &b_bin);
    c = (float *) enif_make_new_binary(env,  n2 * sizeof(float), &c_bin);

   
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n2 * sizeof(float)));
  
    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n2 * sizeof(float), hipMemcpyHostToDevice));
  
    dim3 blocks(in_n,1,1);
    dim3 threads(in_c,1,1);
    pooling_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, st_h, st_w, in_c, in_h, in_w);
  
    // copy to host b,c from GPU dev_b,dev_c
    CHECK(hipMemcpy(b, dev_b, n2 * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(c, dev_c, n2 * sizeof(float), hipMemcpyDeviceToHost));
      

    // return forward data and backward data with tuple {b_bin,c_bin} 
    tuple = enif_make_tuple2(env,b_bin,c_bin);
    
    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

    return(tuple);
}


__global__ void unpooling_kernel(float *a, float *b, float *c, int st_h, int st_w, int in_c, int in_h, int in_w)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1,max_h,max_w,in_h1,in_w1;
    float loss,elt;

    n1 = bid;
    c1 = tid;
    in_h1 = in_h * st_h;
    in_w1 = in_w * st_w;
    for(h2=0;h2<in_h;h2++){
        for(w2=0;w2<in_w;w2++){
            start_h1 = st_h*h2;
            end_h1 = st_h*(h2+1);
            start_w1 = st_w*w2;
            end_w1 = st_w*(w2+1);
            elt = a[IDX4C(n1,c1,h2,w2,in_c,in_h,in_w)];
            loss = b[IDX4C(n1,c1,h2,w2,in_c,in_h,in_w)];
            max_h = (int) floor(elt / 1000.0);
            max_w = (int) fmodf(elt,1000.0);
            for(h1=start_h1;h1<end_h1;h1++){
                for(w1=start_w1;w1<end_w1;w1++){
                    if(h1 == max_h && w1 == max_w){
                        c[IDX4C(n1,c1,h1,w1,in_c,in_h1,in_w1)] = loss;
                    }
                    else{
                        c[IDX4C(n1,c1,h1,w1,in_c,in_h1,in_w1)] = 0.0;
                    }
                }
            }
        }
    }
        
}
  
/*
1st arg in_n of sparse-tensor
2nd arg in_c of sparse-tensor
3rd arg in_h of sparse-tensor
4th arg in_w of sparse-tensor
5th arg binary of sparse-tensor
6th arg binary of loss-tensor
7th arg stride 

return gradiate tensor
e.g.
ts1 index-tensor
  each element is  row*1000+col
  |1.0*1000+1.0,1.0*1000*3.0|
  |3.0*1000+1.0,3.0*1000+3.0|
ts2 loss-tensor
  |0.1,0.2|
  |0.3,0.4|

return
  |0.0,0.0,0.0,0.0|
  |0.0,0.1,0.0,0.2|
  |0.0,0.0,0.0,0.0|
  |0.0,3.4,0.0,0.4|

*/
static ERL_NIF_TERM
unpooling1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  c_bin;
    int in_n,in_c,in_h,in_w,st_h,st_w, n1, n2;
    float *a,*b, *c;
    float *dev_a, *dev_b, *dev_c;
  
    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
    if (!enif_inspect_binary(env, argv[5], &b_bin )) return enif_make_int(env,6);
    if (!enif_get_int(env, argv[6], &st_h)) return enif_make_int(env,7);
    if (!enif_get_int(env, argv[7], &st_w)) return enif_make_int(env,8);

    n1 = in_n * in_c * in_h * in_w;
    n2 = in_n * in_c * (in_h * st_h) * (in_w * st_w);
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env,  n2 * sizeof(float), &c_bin);


      
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n2 * sizeof(float)));

  
    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n2 * sizeof(float), hipMemcpyHostToDevice));
    
    dim3 blocks(in_n,1,1);
    dim3 threads(in_c,1,1);
    unpooling_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, st_h, st_w, in_c, in_h, in_w);
  
    // copy to host d from GPU dev_d
    CHECK(hipMemcpy(c, dev_c, n2 * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return(c_bin);
}

  

__global__ void convolute1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w,
     int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int oh, int ow)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,c2,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1;
    float sum,elt1,elt2;
      
    n1 = bid;
    c2 = tid;
    for(w2=0;w2<ow;w2++){
        for(h2=0;h2<oh;h2++){
            sum = 0.0;
            start_h1 = st_h*h2-pad;
            end_h1 = start_h1 + filt_h;
            start_w1 = st_w*w2-pad;
            end_w1 = start_w1 + filt_w;
            for(c1=0;c1<in_c;c1++){
                for(h1=start_h1;h1<end_h1;h1++){
                    for(w1=start_w1;w1<end_w1;w1++){
                        if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
                            elt1 = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)];
                            elt2 = b[IDX4C(c2,c1,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)];
                            sum = sum + elt1*elt2;
                        }
                    }
                }
            }
            c[IDX4C(n1,c2,h2,w2,filt_n,oh,ow)] = sum;   
        }
    }
        
}
  
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg binary of input tensor
10th arg binary of filter tensor
11th arg stride
12th arg padding   
*/
static ERL_NIF_TERM
convolute1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  c_bin;
    int in_n,in_c,in_h,in_w, filt_n,filt_c,filt_h,filt_w, st_h,st_w,pad, n1, n2, n3, oh, ow;
    float *a,*b, *c;
    float *dev_a, *dev_b, *dev_c;
  
    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &filt_n)) return enif_make_int(env,5);
    if (!enif_get_int(env, argv[5], &filt_c)) return enif_make_int(env,6);
    if (!enif_get_int(env, argv[6], &filt_h)) return enif_make_int(env,7);
    if (!enif_get_int(env, argv[7], &filt_w)) return enif_make_int(env,8);
    if (!enif_inspect_binary(env, argv[8], &a_bin )) return enif_make_int(env,9);
    if (!enif_inspect_binary(env, argv[9], &b_bin )) return enif_make_int(env,10);
    if (!enif_get_int(env, argv[10], &st_h)) return enif_make_int(env,11);
    if (!enif_get_int(env, argv[11], &st_w)) return enif_make_int(env,12);
    if (!enif_get_int(env, argv[12], &pad)) return enif_make_int(env,13);

    
    n1 = in_n * in_c * in_h * in_w;
    n2 = filt_n * filt_c * filt_h * filt_w;
    oh = (in_h+2*pad-filt_h)/st_h + 1;
    ow = (in_w+2*pad-filt_w)/st_w + 1;
    n3 = in_n * filt_n * oh * ow;  // n of filter generate n channel
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env,  n3 * sizeof(float), &c_bin);

    
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

  
    // copy from host a,b,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks(in_n,1,1);
    dim3 threads(filt_n,1,1);
    convolute1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad, in_c, in_h, in_w, oh, ow);
  
    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return(c_bin);
}



__global__ void deconvolute1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w,
     int st_h, int st_w, int pad1, int pad, int in_c, int in_h, int in_w, int oh, int ow, int oh1, int ow1)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,c2,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1;
    float sum,elt1,elt2;
    
    n1 = bid;
    c2 = tid;
        
    //full convolute. stride=1 always
    for(w2=0;w2<ow;w2++){
        for(h2=0;h2<oh;h2++){
            start_h1 = h2-pad1;  
            end_h1 = start_h1 + filt_h;
            start_w1 = w2-pad1;
            end_w1 = start_w1 + filt_w;
            sum = 0.0;
            for(h1=start_h1;h1<end_h1;h1++){
                for(w1=start_w1;w1<end_w1;w1++){
                    for(c1=0;c1<filt_n;c1++){        
                        if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
                            elt1 = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)]; //loss tensor
                            elt2 = b[IDX4C(c1,c2,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)]; //filter tensor
                            sum = sum + elt1*elt2;
                        } 
                    }   
                }
            }
            if(h2-pad >=0 && h2-pad < oh1 && w2-pad >= 0 && w2-pad < ow1){
                c[IDX4C(n1,c2,h2-pad,w2-pad,filt_c,oh1,ow1)] = sum;
            }             
        }
    }
    
}
  
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg binary of input loss tensor
10th arg binary of filter tensor
11th arg stride hight
12th arg stride width
13th arg padding   

memo
ex padding = 1
loss 4*4
filter 2*2
input 3*3  padding=1
(3-2+2*1)/1 + 1 = 4  
decovolute compute 5*5(3*3 padding=1) and save result range 3*3


*/
static ERL_NIF_TERM
deconvolute1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  c_bin;
    int in_n,in_c,in_h,in_w, filt_n,filt_c,filt_h,filt_w, st_h,st_w,pad, pad1, n1, n2, n3, oh, ow, oh1, ow1, i,j,k,l;
    float *a,*b, *b1, *c;
    float *dev_a, *dev_b, *dev_c;
  
    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &filt_n)) return enif_make_int(env,5);
    if (!enif_get_int(env, argv[5], &filt_c)) return enif_make_int(env,6);
    if (!enif_get_int(env, argv[6], &filt_h)) return enif_make_int(env,7);
    if (!enif_get_int(env, argv[7], &filt_w)) return enif_make_int(env,8);
    if (!enif_inspect_binary(env, argv[8], &a_bin )) return enif_make_int(env,9);
    if (!enif_inspect_binary(env, argv[9], &b_bin )) return enif_make_int(env,10);
    if (!enif_get_int(env, argv[10], &st_h)) return enif_make_int(env,11);
    if (!enif_get_int(env, argv[11], &st_w)) return enif_make_int(env,12);
    if (!enif_get_int(env, argv[12], &pad)) return enif_make_int(env,13);

    

    n1 = in_n * in_c * in_h * in_w;
    n2 = filt_n * filt_c * filt_h * filt_w;
    pad1 = filt_h - 1;
    // pad1 = filt_h -1,  pad is original padding size
    oh = (in_h+2*pad1-filt_h)/st_h + 1;
    ow = (in_w+2*pad1-filt_w)/st_w + 1;
    oh1 = (in_h+2*(pad1-pad)-filt_h)/st_h + 1;
    ow1 = (in_w+2*(pad1-pad)-filt_w)/st_w + 1;
    n3 = in_n * filt_c * oh1 * ow1;  // channel of filter generate same channel input tensor
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    b1 = (float *) enif_alloc(n2 * sizeof(float));
    c = (float *) enif_make_new_binary(env,  n3 * sizeof(float), &c_bin);
  
      
    //rotate 180 degree
    for(i=0;i<filt_n;i++){  
        for(j=0;j<filt_c;j++){
            for(k=0;k<filt_h;k++){
                for(l=0;l<filt_w;l++){
                    b1[IDX4C(i,j,filt_h-k-1,filt_w-l-1,filt_c,filt_h,filt_w)] = b[IDX4C(i,j,k,l,filt_c,filt_h,filt_w)];
                }
            }
        }
    }

    
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

  
    // copy from host a,b1,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b1, n2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks(in_n,1,1);
    dim3 threads(filt_c,1,1);
    deconvolute1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad1, pad, in_c, in_h, in_w, oh, ow, oh1, ow1);
  
    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    enif_free(b1);
    
    
    return(c_bin);
}




__global__ void deconvolute2_kernel(float *a1, float *a, float *b, float *c, int filt_n, int filt_c,int filt_h, int filt_w,
     int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int loss_h, int loss_w)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,c2,h1,w1,h2,w2,oh,ow,start_h1,end_h1,start_w1,end_w1;
    int j,k,l,k1,l1;
    float sum,elt1,elt2;
    
    n1 = bid;
    c2 = tid;
    // caution! stride=1 
    oh = (in_h+2*pad-filt_h) + 1;
    ow = (in_w+2*pad-filt_w) + 1;
    
    //dilate loss tensor.
    for(j=0;j<filt_n;j++){
        for(k=0;k<loss_h;k++){
            for(l=0;l<loss_w;l++){
                elt1 = a[IDX4C(n1,j,k,l,in_c,loss_h,loss_w)];
                k1 = st_h*k;
                l1 = st_w*l;
                a1[IDX4C(n1,j,k1,l1,in_c,in_h,in_w)] = elt1;
            }
        }
    }
    //full convulute. stride=1
    for(c2=0;c2<filt_c;c2++){
        for(w2=0;w2<ow;w2++){
            for(h2=0;h2<oh;h2++){
                start_h1 = h2-pad;
                end_h1 = start_h1 + filt_h;
                start_w1 = w2-pad;
                end_w1 = start_w1 + filt_w;
                sum = 0.0;
                for(h1=start_h1;h1<end_h1;h1++){
                    for(w1=start_w1;w1<end_w1;w1++){
                        for(c1=0;c1<filt_n;c1++){        
                            if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
                                elt1 = a1[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)]; //loss tensor
                                elt2 = b[IDX4C(c1,c2,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)]; //filter tensor
                                sum = sum + elt1*elt2;
                            }
                        }
                    }   
                }
                c[IDX4C(n1,c2,h2,w2,filt_c,oh,ow)] = sum;              
            }
        }
    }
    
}




/*
dilate loss tensor 
e.g.

|1.0,2.0|
|3.0,4.0|

dilated stride=2
|1.0,0.0,2.0|
|0.0,0.0,0.0|
|3.0,0.0,4.0|


*/


/*
1st arg in_n of input loss tensor
2nd arg in_c of input loss tensor
3rd arg in_h of input loss  tensor
4th arg in_w of input loss tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg binary of input loss tensor
10th arg binary of filter tensor
11th arg stride hight
12th arg stride width
13th arg padding   
*/
static ERL_NIF_TERM
deconvolute2(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  c_bin;
    int in_n,in_c,in_h,in_w,filt_n,filt_c,filt_h, filt_w, st_h, st_w,pad, pad1, n1, n2, n3, oh, ow, i,j,k,l, loss_h, loss_w;
    float *a, *a1, *b, *b1, *c;
    float *dev_a, *dev_a1, *dev_b, *dev_c;

  
    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &loss_h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &loss_w)) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &filt_n)) return enif_make_int(env,5);
    if (!enif_get_int(env, argv[5], &filt_c)) return enif_make_int(env,6);
    if (!enif_get_int(env, argv[6], &filt_h)) return enif_make_int(env,7);
    if (!enif_get_int(env, argv[7], &filt_w)) return enif_make_int(env,8);
    if (!enif_inspect_binary(env, argv[8], &a_bin )) return enif_make_int(env,9);
    if (!enif_inspect_binary(env, argv[9], &b_bin )) return enif_make_int(env,10);
    if (!enif_get_int(env, argv[10], &st_h)) return enif_make_int(env,11);
    if (!enif_get_int(env, argv[11], &st_w)) return enif_make_int(env,12);
    if (!enif_get_int(env, argv[12], &pad)) return enif_make_int(env,13);

        
    // size for dilate
    in_h = loss_h + (loss_h - 1)*(st_h - 1);
    in_w = loss_w + (loss_w - 1)*(st_w - 1);

    n1 = in_n * in_c * in_h * in_w;  //loss tensor size 
    n2 = filt_n * filt_c * filt_h * filt_w;  //filter tensor size
    pad1 = (filt_h - 1) + pad;    //padding size with dilate
    oh = (in_h+2*pad1-filt_h) + 1; //output deconvolute tensor size. caution stride=1.
    ow = (in_w+2*pad1-filt_w) + 1; // 
    n3 = in_n * filt_c * oh * ow;   // 
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    a1 = (float *) enif_alloc(n1 * sizeof(float));
    b1 = (float *) enif_alloc(n2 * sizeof(float));
    c = (float *) enif_make_new_binary(env,  n3 * sizeof(float), &c_bin);

    //rotate 180 degree
    for(i=0;i<filt_n;i++){  
        for(j=0;j<filt_c;j++){
            for(k=0;k<filt_h;k++){
                for(l=0;l<filt_w;l++){
                    b1[IDX4C(i,j,filt_h-k-1,filt_w-l-1,filt_c,filt_h,filt_w)] = b[IDX4C(i,j,k,l,filt_c,filt_h,filt_w)];
                }
            }
        }
    }


    // dilate 
    for(i=0;i<n1;i++){
        a1[i] = 0.0;
    }

    CHECK(hipMalloc((void**)&dev_a1, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_a, in_n*1*loss_h*loss_w * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

    CHECK(hipMemcpy(dev_a1, a1, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_a, a, in_n*1*loss_h*loss_w  * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b1, n2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks(in_n,1,1);
    dim3 threads(filt_c,1,1);
    deconvolute2_kernel <<<blocks, filt_c>> >(dev_a1, dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad1, in_c, in_h, in_w, loss_h, loss_w);
  
    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
    hipFree(dev_a1);
    hipFree(dev_b);
    hipFree(dev_c);
    enif_free(a1);
    enif_free(b1);
  
    return(c_bin);
}




__global__ void gradfilter1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w, int loss_c, int loss_h, int loss_w, int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int n)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,c2,h1,w1,h2,w2,h3,w3;
    float sum,elt1,elt2;

    n1 = bid;
    c2 = tid;
    
    for(c1=0;c1<filt_c;c1++){
    //h1,w1 is index of filter
        for(h1=0;h1<filt_h;h1++){
            for(w1=0;w1<filt_w;w1++){
                //h2,w2 is index of loss tensor
                sum = 0.0;
                for(h2=0;h2<loss_h;h2++){
                    for(w2=0;w2<loss_w;w2++){
                        //h3,w3 is index of input tensor
                        h3 = h1 - pad + h2;
                        w3 = w1 - pad + w2;
                        if(h3>=0 && h3<in_h && w3>=0 && w3<in_w){
                            elt1 = a[IDX4C(n1,c1,h3,w3,in_c,in_h,in_w)];    //input tensor
                            elt2 = b[IDX4C(n1,c2,h2,w2,loss_c,loss_h,loss_w)]; //loss tensor
                            sum = sum + elt1*elt2;
                        }
                    }
                }
                //set filter tensor
                c[IDX5C(n1,c2,c1,h1,w1,filt_n,filt_c,filt_h,filt_w)] =  sum;
            }
        }
    } 
               
}



  
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg loss_c of loss tensor
10th arg loss_h of loss tensor
11th arg loss_w of loss tensor
12th arg binary of filter tensor
13th arg binary of loss tensor
14th arg stride hight
15th arg stride width
16th arg padding   
*/
static ERL_NIF_TERM
gradfilter1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  c_bin,d_bin;
    int in_n,in_c,in_h,in_w,filt_n,filt_c,filt_h,filt_w,loss_c,loss_h,loss_w,st_h,st_w,pad,n1,n2,n3,n4,i,j,k,l,m;
    float *a,*b,*c,*d;
    float *dev_a, *dev_b, *dev_c;
    float elt;
  
    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &filt_n)) return enif_make_int(env,5);
    if (!enif_get_int(env, argv[5], &filt_c)) return enif_make_int(env,6);
    if (!enif_get_int(env, argv[6], &filt_h)) return enif_make_int(env,7);
    if (!enif_get_int(env, argv[7], &filt_w)) return enif_make_int(env,8);
    if (!enif_get_int(env, argv[8], &loss_c)) return enif_make_int(env,9);
    if (!enif_get_int(env, argv[9], &loss_h)) return enif_make_int(env,10);
    if (!enif_get_int(env, argv[10], &loss_w)) return enif_make_int(env,11);
    if (!enif_inspect_binary(env, argv[11], &a_bin )) return enif_make_int(env,12);
    if (!enif_inspect_binary(env, argv[12], &b_bin )) return enif_make_int(env,13);
    if (!enif_get_int(env, argv[13], &st_h)) return enif_make_int(env,14);
    if (!enif_get_int(env, argv[14], &st_w)) return enif_make_int(env,15);
    if (!enif_get_int(env, argv[15], &pad)) return enif_make_int(env,16);

    n1 = in_n * in_c * in_h * in_w;
    n2 = in_n * loss_c * loss_h * loss_w;
    n3 = in_n * filt_n * filt_c * filt_h * filt_w;
    n4 = filt_n * filt_c * filt_h * filt_w;
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env,  n3 * sizeof(float), &c_bin);
    d = (float *) enif_make_new_binary(env,  n4 * sizeof(float), &d_bin);

    //initialize c
    for(i=0;i<n3;i++){
        c[i] = 0.0;
    }
  
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

    
    // copy from host a,b,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks(in_n,1,1);
    dim3 threads(filt_n,1,1);
    gradfilter1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, loss_c, loss_h, loss_w, st_h, st_w, pad, in_c, in_h, in_w, in_n);
  
    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

    //average
    // clear d
    for(i=0;i<n4;i++){
        d[i] = 0.0;
    }
    // copy from c to d and compute sum
    for(i=0;i<in_n;i++){
        for(j=0;j<filt_n;j++){
            for(k=0;k<filt_c;k++){
                for(l=0;l<filt_h;l++){
                    for(m=0;m<filt_w;m++){
                        elt = c[IDX5C(i,j,k,l,m,filt_n,filt_c,filt_h,filt_w)];
                        d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] = d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] + elt;
                    }
                }
            }
        }
    }
    // average
    for(i=0;i<n4;i++){
        d[i] = d[i] / (float)in_n;
    }
    
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
  
    return(d_bin);
}



__global__ void gradfilter2_kernel(float *a, float *b1, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w, int loss_c, int loss_h, int loss_w, int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int n)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,c2,h1,w1,h2,w2,h3,w3,loss_h1,loss_w1,j,k,l,k1,l1;
    float sum,elt1,elt2;
    
    n1 = bid;
    c2 = tid;
    //dilated loss tensor size
    loss_h1 = loss_h+(loss_h-1)*(st_h-1);
    loss_w1 = loss_w+(loss_w-1)*(st_w-1);
    //dilate loss tensor.
    for(j=0;j<loss_c;j++){
        for(k=0;k<loss_h;k++){
            for(l=0;l<loss_w;l++){
                elt1 = b[IDX4C(n1,j,k,l,loss_c,loss_h,loss_w)];
                k1 = st_h*k;
                l1 = st_w*l;
                b1[IDX4C(n1,j,k1,l1,loss_c,loss_h1,loss_w1)] = elt1;
            }
        }
    }
    //convolute input tensor with dilated loss tensor. cuation stride is always 1. 
    for(c1=0;c1<filt_c;c1++){
    //h1,w1 is index of filter
        for(h1=0;h1<filt_h;h1++){
            for(w1=0;w1<filt_w;w1++){
                //h2,w2 is index of loss tensor
                sum = 0.0;
                for(h2=0;h2<loss_h1;h2++){
                    for(w2=0;w2<loss_w1;w2++){
                        //h3,w3 is index of input tensor
                        h3 = h1 - pad + h2;
                        w3 = w1 - pad + w2;
                        if(h3>=0 && h3<in_h && w3>=0 && w3<in_w){
                            elt1 = a[IDX4C(n1,c1,h3,w3,in_c,in_h,in_w)];    //input tensor
                            elt2 = b1[IDX4C(n1,c2,h2,w2,loss_c,loss_h1,loss_w1)]; //loss tensor
                            sum = sum + elt1*elt2;
                        }
                    }
                }
                //set filter tensor
                c[IDX5C(n1,c2,c1,h1,w1,filt_n,filt_c,filt_h,filt_w)] = + sum;
            }
        }
    } 
        
}

/*
dilate loss tensor 
e.g.

|1.0,2.0|
|3.0,4.0|

dilated stride=2
|1.0,0.0,2.0|
|0.0,0.0,0.0|
|3.0,0.0,4.0|


*/

/*
gradfilter2 is for stride >= 2. This one requires dilate
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg loss_c of loss tensor
10th arg loss_h of loss tensor
11th arg loss_w of loss tensor
12th arg binary of filter tensor
13th arg binary of loss tensor
14th arg stride hight
15th arg stride width
16th arg padding  

*/
static ERL_NIF_TERM
gradfilter2(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  c_bin,d_bin;
    int in_n,in_c,in_h,in_w,filt_n,filt_c,filt_h,filt_w,loss_c,loss_h,loss_w,st_h,st_w,pad,n1,n2,n3,n4,n5,i,j,k,l,m;
    float *a,*b,*b1,*c,*d;
    float *dev_a, *dev_b, *dev_b1, *dev_c;
    float elt;
  
    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &filt_n)) return enif_make_int(env,5);
    if (!enif_get_int(env, argv[5], &filt_c)) return enif_make_int(env,6);
    if (!enif_get_int(env, argv[6], &filt_h)) return enif_make_int(env,7);
    if (!enif_get_int(env, argv[7], &filt_w)) return enif_make_int(env,8);
    if (!enif_get_int(env, argv[8], &loss_c)) return enif_make_int(env,9);
    if (!enif_get_int(env, argv[9], &loss_h)) return enif_make_int(env,10);
    if (!enif_get_int(env, argv[10], &loss_w)) return enif_make_int(env,11);
    if (!enif_inspect_binary(env, argv[11], &a_bin )) return enif_make_int(env,12);
    if (!enif_inspect_binary(env, argv[12], &b_bin )) return enif_make_int(env,13);
    if (!enif_get_int(env, argv[13], &st_h)) return enif_make_int(env,14);
    if (!enif_get_int(env, argv[14], &st_w)) return enif_make_int(env,15);
    if (!enif_get_int(env, argv[15], &pad)) return enif_make_int(env,16);

    n1 = in_n * in_c * in_h * in_w;
    n2 = in_n * loss_c * loss_h * loss_w;
    n3 = in_n * filt_n * filt_c * filt_h * filt_w;
    n4 = filt_n * filt_c * filt_h * filt_w;
    n5 = in_n * loss_c * (loss_h+(loss_h-1)*(st_h-1)) * (loss_w+(loss_w-1)*(st_w-1));  // dilated loss tensor size  
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    b1 = (float *) enif_alloc(n5 * sizeof(float)); // dilate loss tensor area
    c = (float *) enif_make_new_binary(env,  n3 * sizeof(float), &c_bin);
    d = (float *) enif_make_new_binary(env,  n4 * sizeof(float), &d_bin);

    //initialize c
    for(i=0;i<n3;i++){
        c[i] = 0.0;
    }
    //initialize b1
    for(i=0;i<n5;i++){
        b1[i] = 0.0;
    }
  
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b1, n5 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

    
    // copy from host a,b,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b1, b1, n5 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks(in_n,1,1);
    dim3 threads(filt_n,1,1);
    gradfilter2_kernel <<<blocks, threads>> >(dev_a, dev_b1, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, loss_c, loss_h, loss_w, st_h, st_w, pad, in_c, in_h, in_w, in_n);
  
    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

    //average
    // clear d
    for(i=0;i<n4;i++){
        d[i] = 0.0;
    }
    // copy from c to d and compute sum
    for(i=0;i<in_n;i++){
        for(j=0;j<filt_n;j++){
            for(k=0;k<filt_c;k++){
                for(l=0;l<filt_h;l++){
                    for(m=0;m<filt_w;m++){
                        elt = c[IDX5C(i,j,k,l,m,filt_n,filt_c,filt_h,filt_w)];
                        d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] = d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] + elt;
                    }
                }
            }
        }
    }
    // average
    for(i=0;i<n4;i++){
        d[i] = d[i] / (float)in_n;
    }
     
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_b1);
    hipFree(dev_c);
    enif_free(b1);
    return(d_bin);
}



__global__ void full_kernel(float *a, float *b, int in_n, int in_c, int in_h, int in_w, int n)
{
    int tid = threadIdx.x;
    int n1,i,j,k;
    float elt;
    if(tid < n)
    {   
        n1 = tid;
        for(i=0;i<in_c;i++){
            for(j=0;j<in_h;j++){
                for(k=0;k<in_w;k++){
                    elt = a[IDX4C(n1,i,j,k,in_c,in_h,in_w)];
                    b[IDX2C(n1,i*in_h*in_w + j*in_w + k,in_n)] = elt;
                }
            }
        }
    }
}
  
/*
1st arg in_n of input tensor 4DIM
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg binary of input tensor
*/
static ERL_NIF_TERM
full1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int in_n,in_c,in_h,in_w,n1,n;
    float *a,*b;
    float *dev_a, *dev_b;

    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);

 
    n1 = in_n * in_c * in_h * in_w;
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env,  n1 * sizeof(float), &b_bin);
    n = in_n;
      
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n1 * sizeof(float)));
  
    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n1 * sizeof(float), hipMemcpyHostToDevice));

    full_kernel << <1, n>> >(dev_a, dev_b, in_n, in_c, in_h, in_w, n);
  
    // copy to host d from GPU dev_d
    CHECK(hipMemcpy(b, dev_b, n1 * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
  
    return(b_bin);
}


__global__ void unfull_kernel(float *a, float *b, int in_n, int in_c, int in_h, int in_w, int n)
{
    int tid = threadIdx.x;
    int n1,i,j,k;
    float elt;
    if(tid < n)
    {   
        n1 = tid;
        for(i=0;i<in_c;i++){
            for(j=0;j<in_h;j++){
                for(k=0;k<in_w;k++){
                    elt = a[IDX2C(n1,i*in_h*in_w + j*in_w + k,in_n)];
                    b[IDX4C(n1,i,j,k,in_c,in_h,in_w)] = elt;
                }
            }
        }
    }
}
  
/*
1st arg in_n of input tensor 4DIM
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg binary of input tensor
*/
static ERL_NIF_TERM
unfull1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int in_n,in_c,in_h,in_w,n1,n;
    float *a,*b;
    float *dev_a, *dev_b;
    
    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);

    n1 = in_n * in_c * in_h * in_w;
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env,  n1 * sizeof(float), &b_bin);
    n = in_n;
      
      // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n1 * sizeof(float)));
  
    // copy from host a,b1,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n1 * sizeof(float), hipMemcpyHostToDevice));

    unfull_kernel << <1, n>> >(dev_a, dev_b, in_n, in_c, in_h, in_w, n);
  
    // copy to host d from GPU dev_d
    CHECK(hipMemcpy(b, dev_b, n1 * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
  
    return(b_bin);
}


static ERL_NIF_TERM
new1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    int n,i;
    ERL_NIF_TERM a_bin;
    float *a;
    double d;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_get_double(env, argv[1], &d)) return enif_make_int(env,2);
    a = (float *) enif_make_new_binary(env, n * sizeof(float), &a_bin);

    // Set matrix data 
    for(i=0;i<n;i++){
        a[i] = (float)d;
    }

    return(a_bin);
}



static ERL_NIF_TERM
new2(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    int r1,c1,i,j;
    ERL_NIF_TERM head, list, a_bin;
    float *a;
    double d;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    a = (float *) enif_make_new_binary(env, r1 * c1 * sizeof(float), &a_bin);

    // Set matrix data 
    list = argv[2]; /* matrix1 */
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            enif_get_list_cell(env, list, &head, &list);
            enif_get_double(env,head,&d);
            a[IDX2C(i,j,r1)] = (float)d;
        }
    }

    return(a_bin);
}


static ERL_NIF_TERM
new3(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    int c,h,w,i,j,k;
    ERL_NIF_TERM head, list, a_bin;
    float *a;
    double d;

    if (!enif_get_int(env, argv[0], &c)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &h)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &w)) return enif_make_int(env,3);
    a = (float *) enif_make_new_binary(env, c * h * w *  sizeof(float), &a_bin);

    // Set matrix data 
    list = argv[3]; /* matrix1 */
    for(i=0;i<c;i++){
        for(j=0;j<h;j++){
            for(k=0;k<w;k++){
                enif_get_list_cell(env, list, &head, &list);
                enif_get_double(env,head,&d);
                a[IDX3C(i,j,k,h,w)] = (float)d;
            }
        }
    }

    return(a_bin);
}



static ERL_NIF_TERM
new4(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    int n,c,h,w,i,j,k,l;
    ERL_NIF_TERM head, list, a_bin;
    float *a;
    double d;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &w)) return enif_make_int(env,4);
    a = (float *) enif_make_new_binary(env, n * c * h * w *  sizeof(float), &a_bin);

    // Set matrix data 
    list = argv[4]; /* matrix1 */
    for(i=0;i<n;i++){
        for(j=0;j<c;j++){
            for(k=0;k<h;k++){
                for(l=0;l<w;l++){
                    enif_get_list_cell(env, list, &head, &list);
                    enif_get_double(env,head,&d);
                    a[IDX4C(i,j,k,l,c,h,w)] = (float)d;
                }
            }
        }
    }

    return(a_bin);
}



static ERL_NIF_TERM
rand1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    int n,i;
    float x,y,val;
    float *result_data;
    ERL_NIF_TERM result;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    result_data = (float *) enif_make_new_binary(env, n * sizeof(float), &result);

    srand((unsigned) time(NULL));
    for(i=0;i<n;i++){
        //box_muller
        x = (float)rand()/(float)RAND_MAX;
        y = (float)rand()/(float)RAND_MAX;
        val = sqrt(-2.0 * log(x)) * cos(2.0 * PI * y);
        result_data[i] = val;
    }
    return(result);
}



static ERL_NIF_TERM
mult1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin, b_bin;
    ERL_NIF_TERM  c_bin;
    int r1, c1, r2, c2, n, i, j;
    float *a,*b,*c;
    float* devPtrA;
    float* devPtrB;
    float* devPtrC;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &r2)) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &c2)) return enif_make_int(env,5);
    if (!enif_inspect_binary(env, argv[5], &b_bin)) return enif_make_int(env,6);
    n = r1*c2;
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);

    for(j=0;j<c2;j++)
        for(i=0;i<r1;i++)
            c[IDX2C(i,j,r1)] = 0.0;


    // Initialize CUBLAS
    cublasInit();

    CUBLAS(cublasAlloc (r1*c1, sizeof(*a), (void**)&devPtrA));
    CUBLAS(cublasAlloc (r2*c2, sizeof(*b), (void**)&devPtrB));
    CUBLAS(cublasAlloc (r1*c2, sizeof(*c), (void**)&devPtrC));

    CUBLAS(hipblasSetMatrix (r1, c1, sizeof(*a), a, r1, devPtrA, r1));
    CUBLAS(hipblasSetMatrix (r2, c2, sizeof(*b), b, r2, devPtrB, r2));
    CUBLAS(hipblasSetMatrix (r1, c2, sizeof(*c), c, r1, devPtrC, r1));


    //Sgemm
    hipblasSgemm('N', 'N', r1, c2, c1, 1.0, devPtrA, r1, devPtrB, r2, 0.0, devPtrC, r1);


    CUBLAS(hipblasGetMatrix (r1, c2, sizeof(*c), devPtrC, r1, c, r1));
    
    // Shutdown CUBLAS
    cublasFree(devPtrA);
    cublasFree(devPtrB);
    cublasFree(devPtrC);
    cublasShutdown();
    

    return(c_bin);

}


__global__ void add1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}



static ERL_NIF_TERM
add1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin, b_bin;
    ERL_NIF_TERM  c_bin;
    int n;
    float *a,*b,*c;
    float *dev_a, *dev_b, *dev_c;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);


    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);

    // Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	add1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

    return(c_bin);
}

__global__ void sub1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] - b[tid];
		tid += blockDim.x * gridDim.x;
	}
}
static ERL_NIF_TERM
sub1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin, b_bin;
    ERL_NIF_TERM  c_bin;
    int n;
    float *a,*b,*c;
    float *dev_a, *dev_b, *dev_c;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);

    	// Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	sub1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

    return(c_bin);
}




__global__ void emult1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}
}


static ERL_NIF_TERM
emult1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin, b_bin;
    ERL_NIF_TERM  c_bin;
    int n;
    float *a,*b,*c;
    float *dev_a, *dev_b, *dev_c;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
    
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);

    	// Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	emult1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

    return(c_bin);
}


__global__ void ediv1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] / b[tid];
		tid += blockDim.x * gridDim.x;
	}
}


static ERL_NIF_TERM
ediv1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin, b_bin;
    ERL_NIF_TERM  c_bin;
    int n;
    float *a,*b,*c;
    float *dev_a, *dev_b, *dev_c;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
    
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);

    	// Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	ediv1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

    return(c_bin);
}



static ERL_NIF_TERM
transpose1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int r1, c1, n, i, j;
    float *a,*b;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    n = r1*c1;
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);

    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            b[IDX2C(j,i,c1)] = a[IDX2C(i,j,r1)];
        }
    }

    return(b_bin);
}


static ERL_NIF_TERM
ident1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    int n,i,j;
    ERL_NIF_TERM a_bin;
    float *a;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    a = (float *) enif_make_new_binary(env, n * n * sizeof(float), &a_bin);

    // Set matrix data 
    for(i=0;i<n;i++){
        for(j=0;j<n;j++){
            if(i==j)
                a[IDX2C(i,j,n)] = 1.0;
            else
                a[IDX2C(i,j,n)] = 0.0;
        }
    }

    return(a_bin);
}




__global__ void sigmoid_kernel(float *a, float *b, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{   
        b[tid] = SIGMOID(a[tid]);
		tid += blockDim.x * gridDim.x;
	}
}

static ERL_NIF_TERM
activate_sigmoid(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int n;
    float *a,*b;
    float *dev_a, *dev_b;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);

    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);

    	// Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	sigmoid_kernel << <128, 128 >> >(dev_a, dev_b, n);

	// copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);

    return(b_bin);
}



__global__ void tanh_kernel(float *a, float *b, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		b[tid] = tanh(a[tid]);
		tid += blockDim.x * gridDim.x;
	}
}


static ERL_NIF_TERM
activate_tanh(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int n;
    float *a,*b;
    float *dev_a, *dev_b;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);

    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);

    	// Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	tanh_kernel << <128, 128 >> >(dev_a, dev_b, n);

	// copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);

    return(b_bin);
}



__global__ void relu_kernel(float *a, float *b, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{   
        if(a[tid] >= 0)
		    b[tid] = a[tid];
        else 
            b[tid] = 0.0;
		tid += blockDim.x * gridDim.x;
	}
}


static ERL_NIF_TERM
activate_relu(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int n;
    float *a,*b;
    float *dev_a, *dev_b;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,3);

    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);

    	// Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	relu_kernel << <128, 128 >> >(dev_a, dev_b, n);

	// copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);

    return(b_bin);
}

static ERL_NIF_TERM
activate_softmax(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int r1, c1, n, i, j, k;
    float *a,*b;
    float max,sum,delta;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    n = r1*c1;
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);

    //calculate softmax
    delta = 0.01;
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            max = -3.402823e38;
            for(k=0;k<c1;k++){
                if(a[IDX2C(i,k,r1)] > max)
                    max = a[IDX2C(i,k,r1)];
            }
            sum = 0.0;
            for(k=0;k<c1;k++){
                sum = sum + exp(a[IDX2C(i,k,r1)] - max);
            }
            b[IDX2C(i,j,r1)] = exp(a[IDX2C(i,j,r1)] - max) / (sum+delta);
            
        }
    }


    return(b_bin);
}



__global__ void differ_sigmoid_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{   
        
		c[tid] = a[tid] * ((1 - SIGMOID(b[tid])) * SIGMOID(b[tid]));
		tid += blockDim.x * gridDim.x;
	}
}

static ERL_NIF_TERM
differ_sigmoid(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin, b_bin;
    ERL_NIF_TERM  c_bin;
    int n;
    float *a,*b,*c;
    float *dev_a, *dev_b, *dev_c;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);

    	// Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	differ_sigmoid_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

    return(c_bin);
}


__global__ void differ_tanh_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{   
        c[tid] = a[tid] * (1/(cosh(b[tid]) * cosh(b[tid])));
		tid += blockDim.x * gridDim.x;
	}
}

static ERL_NIF_TERM
differ_tanh(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin, b_bin;
    ERL_NIF_TERM  c_bin;
    int n;
    float *a,*b,*c;
    float *dev_a, *dev_b, *dev_c;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
    
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);

    	// Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	differ_tanh_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

    return(c_bin);
}



__global__ void differ_relu_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{   
        if(b[tid] >= 0)
		    c[tid] = a[tid];
        else 
            c[tid] = 0.0;
		tid += blockDim.x * gridDim.x;
	}
}

static ERL_NIF_TERM
differ_relu(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin, b_bin;
    ERL_NIF_TERM  c_bin;
    int n;
    float *a,*b,*c;
    float *dev_a, *dev_b, *dev_c;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,4);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);

    	// Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	differ_relu_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

    return(c_bin);
}


__global__ void smult_kernel(float d, float *a, float *b, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		b[tid] = d * a[tid];
		tid += blockDim.x * gridDim.x;
	}
}


static ERL_NIF_TERM
smult1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int n;
    float *a,*b;
    float *dev_a, *dev_b;
    double s;

    if (!enif_get_double(env, argv[0], &s)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &n)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);

    // Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	smult_kernel << <128, 128 >> >((float)s,dev_a, dev_b, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
	hipFree(dev_b);

    return(b_bin);
}


static ERL_NIF_TERM
trace1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  result;
    int r1, c1, i, j;
    float *a;
    float trace;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    a = (float *) a_bin.data;
    
    trace = 0.0;
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            if(i==j)
                trace = trace + a[IDX2C(i,j,r1)];
        }
    }

    result = enif_make_double(env,trace);

    return(result);
}


static ERL_NIF_TERM
mean_square(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  result;
    int r1, c1, i, j;
    float *a, *b;
    float d,s;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[3], &b_bin )) return enif_make_int(env,4);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;

    s = 0.0;
    for(i=0;i<r1;i++){
        for (j=0;j<c1;j++){
            d = a[IDX2C(i,j,r1)] -  b[IDX2C(i,j,r1)];
            s = s + d*d;            
        }
    } 
    s = s / (2.0*(float(r1)));
    result = enif_make_double(env,s);
    return(result);
}

static ERL_NIF_TERM
cross_entropy(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  result;
    int r1, c1, i, j;
    float *a, *b;
    float d,s,delta;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[3], &b_bin )) return enif_make_int(env,4);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;

    
    delta = 1e-7;
    s = 0.0;
    for(i=0;i<r1;i++){
        for (j=0;j<c1;j++){
            d = a[IDX2C(i,j,r1)] + delta;
            s = s + b[IDX2C(i,j,r1)] * log(d);
        }
    }
    s = -1.0 * s / (float)r1;
    result = enif_make_double(env,s);
    return(result);
}





static ERL_NIF_TERM
elt1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  result;
    int r1, c1, i, j;
    float *a;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &i)) enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &j)) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
    a = (float *) a_bin.data;
    
    result = enif_make_double(env,(double)a[IDX2C(i,j,r1)]);

    return(result);
}

static ERL_NIF_TERM
set1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int r1, c1, n, i, j, x, y;
    float *a,*b;
    double val;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &x)) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &y)) return enif_make_int(env,5);
    if (!enif_get_double(env, argv[5], &val)) return enif_make_int(env,6);


    n = r1*c1;
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);

    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            if(i==x && j==y)
                b[IDX2C(i,j,r1)] = (float)val;
            else 
                b[IDX2C(i,j,r1)] = a[IDX2C(i,j,r1)];
        }
    }


    return(b_bin);
}

static ERL_NIF_TERM
add_diff1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int r1, c1, n, i, j, x, y;
    float *a,*b;
    double val;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &x)) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &y)) return enif_make_int(env,5);
    if (!enif_get_double(env, argv[5], &val)) return enif_make_int(env,6);


    n = r1*c1;
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);

    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            if(i==x && j==y)
                b[IDX2C(i,j,r1)] = a[IDX2C(i,j,r1)] + (float)val;
            else 
                b[IDX2C(i,j,r1)] = a[IDX2C(i,j,r1)];
        }
    }


    return(b_bin);
}

static ERL_NIF_TERM
add_diff2(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int n1, c1, h1, w1, n, i, j, k, l, n2, c2, h2, w2;
    float *a,*b;
    double val;

    if (!enif_get_int(env, argv[0], &n1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &h1)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &w1)) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
    if (!enif_get_int(env, argv[5], &n2)) return enif_make_int(env,6);
    if (!enif_get_int(env, argv[6], &c2)) return enif_make_int(env,7);
    if (!enif_get_int(env, argv[7], &h2)) return enif_make_int(env,8);
    if (!enif_get_int(env, argv[8], &w2)) return enif_make_int(env,9);
    if (!enif_get_double(env, argv[9], &val)) return enif_make_int(env,10);


    n = n1*c1*h1*w1;
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);

    
    for(i=0;i<n1;i++){
        for(j=0;j<c1;j++){
            for(k=0;k<h1;k++){
                for(l=0;l<w1;l++){
                    if(i==n2 && j==c2 && k==h2 && l==w2){
                        b[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(i,j,k,l,c1,h1,w1)] + (float)val;
                    }
                    else {
                        b[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(i,j,k,l,c1,h1,w1)];
                    }
                }
            }
        }
    }


    return(b_bin);
}



static ERL_NIF_TERM
average1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int r1, c1, i, j;
    float *a,*b;
    float sum;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);

    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, c1 * sizeof(float), &b_bin);

    for(j=0;j<c1;j++){
        sum = 0.0;
        for(i=0;i<r1;i++){
            sum = sum + a[IDX2C(i,j,r1)];
        }
        b[j] = sum / (float)r1;
    }


    return(b_bin);
}

/*
1st arg row-size of matrix
2nd arg col-size of matrix
3rd arg matrix data binary
*/


static ERL_NIF_TERM
sum1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  result;
    int r1, c1, i, j;
    float *a;
    float sum;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    a = (float *) a_bin.data;
    
    sum = 0.0;
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            sum = sum + a[IDX2C(i,j,r1)];
        }
    }

    result = enif_make_double(env,sum);

    return(result);
}

/*
transfer 2 DIm matrix to list 
*/
static ERL_NIF_TERM
to_list1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  head,list;
    int r1, c1, i, j;
    float *a;

    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    a = (float *) a_bin.data;

    
    list = enif_make_list(env, 0);
    for(i=r1-1;i>=0;i--){
        for(j=c1-1;j>=0;j--){
            head = enif_make_double(env,(double)a[IDX2C(i,j,r1)]);
            list = enif_make_list_cell(env,head,list);
        }
    }

    return(list);
}
/*
transfer 3 DIm matrix to list
*/

static ERL_NIF_TERM
to_list2(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  head,list;
    int c, h, w, i, j, k;
    float *a;

    if (!enif_get_int(env, argv[0], &c)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &h)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &w)) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[3], &a_bin )) return enif_make_int(env,4);
   
    a = (float *) a_bin.data;
    
    list = enif_make_list(env, 0);
    for(i=c-1;i>=0;i--){
        for(j=h-1;j>=0;j--){
            for(k=w-1;k>=0;k--){
                head = enif_make_double(env,(double)a[IDX3C(i,j,k,h,w)]);
                list = enif_make_list_cell(env,head,list);
            }
        }
    }

    return(list);
}
/*
transfer 4 DIm matrix to list
*/
static ERL_NIF_TERM
to_list3(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  head,list;
    int n, c, h, w, i, j, k, l;
    float *a;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &w)) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_badarg(env);
    a = (float *) a_bin.data;

    
    list = enif_make_list(env, 0);
    for(i=n-1;i>=0;i--){
        for(j=c-1;j>=0;j--){
            for(k=h-1;k>=0;k--){
                for(l=w-1;l>=0;l--){
                    head = enif_make_double(env,(double)a[IDX4C(i,j,k,l,c,h,w)]);
                    list = enif_make_list_cell(env,head,list);
                }
            }
        }
    }

    return(list);
}

__global__ void dropout1_kernel(float *a, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
        a[tid] = 1.0;
		tid += blockDim.x * gridDim.x;
	}
}

/*
1st arg size of mask tensor
2nd arg rate of dropout

return mask tensor
element of mask tensor is basicaly 1.0.
element of dropout rate is 0.0.
when forward and backward, generate Hadamard product with mask tensor
*/
static ERL_NIF_TERM
dropout1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ERL_NIF_TERM  a_bin;
    int n,count,i,j;
    float *a,*dev_a;
    double dropout_rate;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_get_double(env, argv[1], &dropout_rate)) return enif_make_int(env,2);

    a = (float *) enif_make_new_binary(env, n * sizeof(float), &a_bin);

    // Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));

    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));

	dropout1_kernel << <128, 128 >> >(dev_a, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(a, dev_a, n * sizeof(float), hipMemcpyDeviceToHost));


    // dropout
    count = (int)(double(n)*dropout_rate);
    for(i=0;i<count;i++){
        j = rand() % n;
        a[j] = 0.0;
    }

    // free 
    hipFree(dev_a);

    return(a_bin);
}


__global__ void sgd1_kernel(float *a, float *b, float *c, float lr, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
        c[tid] = a[tid] - b[tid]*lr;
		tid += blockDim.x * gridDim.x;
	}
}
/*
w - g*lr
w is weight matrix.
g is gradient matrix.
when element of w is zero result is zero. This means dropout.
return updated weight matrix.

1st arg is size of vectorized matrix
2nd arg is weight matrix or tensor
3rd arg is gradient matrix or tensor
4th arg is learning rate
*/
static ERL_NIF_TERM
sgd1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin, b_bin;
    ERL_NIF_TERM  c_bin;
    int n;
    float *a,*b,*c,*dev_a, *dev_b, *dev_c;
    float lr;
    double learning_rate;

    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
    if (!enif_get_double(env, argv[3], &learning_rate)) return enif_make_int(env,4);


    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);
    lr = (float) learning_rate;

    	// Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	sgd1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, lr, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));


    // free 
    hipFree(dev_a);
	hipFree(dev_b);    
	hipFree(dev_c);

    return(c_bin);
}


/*
  def momentum(v, g, lr) do
    Matrex.apply(v, g, fn v, g -> 0.5 * v - lr * g end)
  end
*/
__global__ void momentum_kernel(float *a, float *b, float *c, float *d, float *e, float lr, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        
        d[tid] = ((0.9 * b[tid]) - (lr * c[tid]));
        e[tid] = a[tid] + d[tid];
        
        tid += blockDim.x * gridDim.x;
    }
}

/*
1st arg row-size of vectorized each-matrix
2nd arg wight-matrix    (a)
3rd arg v-matrix        (b)
4th arg gradient-matrix (c)
5th arg learning rate

return tuple {next_v-mattrix,weight_matrix}
*/
static ERL_NIF_TERM
momentum1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin,c_bin;
    ERL_NIF_TERM  d_bin,e_bin,tuple;
    int n;
    float *a,*b,*c,*d,*e;
    float *dev_a, *dev_b, *dev_c ,*dev_d, *dev_e;
    float lr;
    double learning_rate;
  
    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[3], &c_bin )) return enif_make_int(env,4);
    if (!enif_get_double(env, argv[4], &learning_rate)) return enif_make_int(env,5);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) c_bin.data;
    d = (float *) enif_make_new_binary(env, n * sizeof(float), &d_bin);
    e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
    lr = (float) learning_rate;
    
  
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
  
    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
  
    momentum_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, lr, n);
  
    // copy to host d from GPU dev_d
    CHECK(hipMemcpy(d, dev_d, n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));

    
    // free 
    hipFree(dev_a);
	hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_d);
	hipFree(dev_e);
    
    tuple = enif_make_tuple2(env,d_bin,e_bin);
    return(tuple);
}

/* ADAGRAD
    h1 = h + grad*grad
    lr1 = lr/(sqrt(h1))
    w1 = w - lr1 * grad 

    a[] = w
    b[] = h
    c[] = grad
    d[] = h1
    e[] = w1
*/
  
__global__ void adagrad_kernel(float *a, float *b, float *c, float *d, float *e, float lr, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float lr1;
    while (tid < n)
    {   
        d[tid] = b[tid] + c[tid]*c[tid];
        if(d[tid] != 0.0)
            lr1 = lr/(sqrt(d[tid]));
        else
            lr1 = lr;
        e[tid] = a[tid] - lr1 * c[tid];

        tid += blockDim.x * gridDim.x;
    }
}
 
/*
1st arg row-size of vectorized each-matrix
2nd arg wight-matrix (a_bin)
3rd arg h-matrix     (b_bin)
4th arg grad-matrix  (c_bin)
5th arg learning rate
return tuple {new-h,new-w}
*/
static ERL_NIF_TERM
adagrad1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin,c_bin;
    ERL_NIF_TERM  d_bin,e_bin,tuple;
    int n;
    float *a,*b,*c,*d,*e;
    float *dev_a, *dev_b, *dev_c, *dev_d, *dev_e;
    float lr;
    double learning_rate;
    
    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[3], &c_bin)) return enif_make_int(env,4);
    if (!enif_get_double(env, argv[4], &learning_rate)) return enif_make_int(env,5);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) c_bin.data;
    d = (float *) enif_make_new_binary(env, n * sizeof(float), &d_bin);
    e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
    lr = (float) learning_rate;
  
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
  
    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
  
    adagrad_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, lr, n);
  
    // copy to host d,e from GPU dev_d,dev_e
    CHECK(hipMemcpy(d, dev_d, n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));

    

    // free 
    hipFree(dev_a);
	hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_d);
	hipFree(dev_e);
    
    tuple = enif_make_tuple2(env,d_bin,e_bin);
    return(tuple);
}

/* RMSprop
    h1 = alpha * h + (1 - alpha) * grad*grad
    lr1 = lr /(sqrt(h) + epsilon)
    w1 = w - lr1 * grad 

    a[] = w
    b[] = h
    c[] = grad
    d[] = h1
    e[] = w1
*/
  
__global__ void rms_kernel(float *a, float *b, float *c, float *d, float *e, float lr, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float lr1,alpha,epsilon;
    alpha = 0.99;
    epsilon = 10.0e-7;
    while (tid < n)
    {   
        d[tid] = alpha * b[tid] + (1-alpha)*c[tid]*c[tid];
        lr1 = lr/(sqrt(d[tid])+epsilon);
        e[tid] = a[tid] - lr1*c[tid];

        tid += blockDim.x * gridDim.x;
    }
}
 
/*
1st arg row-size of vectorized each-matrix
2nd arg wight-matrix (a_bin)
3rd arg h-matrix     (b_bin)
4th arg grad-matrix  (c_bin)
5th arg learning rate
return tuple {new-h,new-w}
*/
static ERL_NIF_TERM
rms1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin,c_bin;
    ERL_NIF_TERM  d_bin,e_bin,tuple;
    int n;
    float *a,*b,*c,*d,*e;
    float *dev_a, *dev_b, *dev_c, *dev_d, *dev_e;
    float lr;
    double learning_rate;
    
    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[3], &c_bin)) return enif_make_int(env,4);
    if (!enif_get_double(env, argv[4], &learning_rate)) return enif_make_int(env,5);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) c_bin.data;
    d = (float *) enif_make_new_binary(env, n * sizeof(float), &d_bin);
    e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
    lr = (float) learning_rate;
  
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
  
    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
  
    rms_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, lr, n);
  
    // copy to host d,e from GPU dev_d,dev_e
    CHECK(hipMemcpy(d, dev_d, n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));

    

    // free 
    hipFree(dev_a);
	hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_d);
	hipFree(dev_e);
    
    tuple = enif_make_tuple2(env,d_bin,e_bin);
    return(tuple);
}


/* ADAM
    beta1 = 0.9
    beta2 = 0.999
    epsilon = 10.0e-7
    alpha = 0.001
    m1 = beta1 * m + (1 - beta1) * grad
    v1 = beta2 * v + (1 - beta2) * grad^2
    m2 = m1/(1 - beta1)
    v2 = v1/(1 - beta2)
    w1 = w - alpha * m2/(sqrt(v2)+epsilon)

    a[] is w
    b[] is m
    c[] is v
    d[] is grad
    e[] is m1
    f[] is v1
    g[] is w1
 */
  
__global__ void adam_kernel(float *a, float *b, float *c, float *d, float *e, float *f, float *g, float lr, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float beta1,beta2,epsilon,m2,v2;
    beta1 = 0.9;
    beta2 = 0.999;
    epsilon = 10.0e-7;
    //alpha = 0.001;

    while (tid < n){   
        e[tid] = beta1 * b[tid] + (1 - beta1) * d[tid];
        f[tid] = beta2 * c[tid] + (1 - beta2) * d[tid]*d[tid];
        m2 = e[tid]/(1-beta1);
        v2 = f[tid]/(1-beta2);
        g[tid] = a[tid] - lr * (m2/(sqrt(v2)+epsilon));
         
        tid += blockDim.x * gridDim.x;
    }
}
   
/*
1st arg row-size of vectorized each-matrix
2nd arg w-matrix     (a_bin)
3rd arg m-matrix     (b_bin)
4th arg v-matrix     (c_bin)
5th arg grad-matrix  (d_bin)
6th arg learning rate

return tuple {m1,v1,w1}
*/
static ERL_NIF_TERM
adam1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin,c_bin,d_bin;
    ERL_NIF_TERM  e_bin,f_bin,g_bin,tuple;
    int n;
    float *a,*b,*c,*d,*e,*f,*g;
    float *dev_a, *dev_b, *dev_c, *dev_d, *dev_e, *dev_f, *dev_g;
    float lr;
    double learning_rate;
      
    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[3], &c_bin)) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &d_bin)) return enif_make_int(env,5);
    if (!enif_get_double(env, argv[5], &learning_rate)) return enif_make_int(env,6);
  
    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) c_bin.data;
    d = (float *) d_bin.data;
    e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
    f = (float *) enif_make_new_binary(env, n * sizeof(float), &f_bin);
    g = (float *) enif_make_new_binary(env, n * sizeof(float), &g_bin);
    lr = (float) learning_rate;
    
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_f, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_g, n * sizeof(float)));

    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_f, f, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_g, g, n * sizeof(float), hipMemcpyHostToDevice));
    
    adam_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, dev_f, dev_g, lr, n);
    
    // copy to host d,e from GPU dev_d,dev_e
    CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(f, dev_f, n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(g, dev_g, n * sizeof(float), hipMemcpyDeviceToHost));
      
  
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_d);
    hipFree(dev_e);
    hipFree(dev_f);
    hipFree(dev_g);
    
      
    tuple = enif_make_tuple3(env,e_bin,f_bin,g_bin);
    return(tuple);
}
  
  


/*
1st arg row-size of matrix
2nd arg col-size of matris
3rd arg predicted matrix
4th arg list of label. each element is integer
return accuracy rate
*/

static ERL_NIF_TERM
accuracy1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  head,list,result;
    int r1, c1, i, j, n, index,sum;
    float *a;
    double max,rate;
  
    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);

    a = (float *) a_bin.data;
    

    // calculate accuracy
    sum = 0;
    list = argv[3]; 
    for(i=0;i<r1;i++){
        max = 0.0;
        enif_get_list_cell(env, list, &head, &list);
        enif_get_int(env,head,&n);
        for(j=0;j<c1;j++){
            if(a[IDX2C(i,j,r1)] > max){
                max = a[IDX2C(i,j,r1)];
                index = j;
            }
        }
        if(index == n)
            sum++;
    }
    rate = (double)sum / (double)r1;
    result = enif_make_double(env,rate);
    return(result);
}

/*
1st arg row-size of matrix
2nd arg col-size of matris
3rd arg predicted matrix
4th arg list of label. each element is integer
return correct number
*/

static ERL_NIF_TERM
correct1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  head,list,result;
    int r1, c1, i, j, n, index,sum;
    float *a;
    float max;
  
    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);

    a = (float *) a_bin.data;
    

    // calculate correct number
    sum = 0;
    list = argv[3]; 
    for(i=0;i<r1;i++){
        max = 0.0;
        enif_get_list_cell(env, list, &head, &list);
        enif_get_int(env,head,&n);
        for(j=0;j<c1;j++){
            if(a[IDX2C(i,j,r1)] > max){
                max = a[IDX2C(i,j,r1)];
                index = j;
            }
        }
        if(index == n)
            sum++;
    }

    result = enif_make_double(env,(double)sum);
    return(result);
}


/*
random_select for matrix data
*/
static ERL_NIF_TERM
random_select1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  c_bin,d_bin,tuple;
    int r1, c1, r2, c2, i, j, n, r;
    float *a, *b, *c, *d;
  
    if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &r2)) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &c2)) return enif_make_int(env,5);
    if (!enif_inspect_binary(env, argv[5], &b_bin )) return enif_make_int(env,6);
    if (!enif_get_int(env, argv[6], &n)) return enif_make_int(env,7);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n*c1 * sizeof(float), &c_bin);
    d = (float *) enif_make_new_binary(env, n*c2 * sizeof(float), &d_bin);


    // random-select
    for(i=0;i<n;i++){
        r = rand() % r1;
        for(j=0;j<c1;j++){
            c[IDX2C(i,j,n)] = a[IDX2C(r,j,r1)];
        }
        for(j=0;j<c2;j++){
            d[IDX2C(i,j,n)] = b[IDX2C(r,j,r2)];
        }    
    }

    tuple = enif_make_tuple2(env,c_bin,d_bin);
    return(tuple);
}

/*
random_select for 4D-tensor data
*/
static ERL_NIF_TERM
random_select2(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  c_bin,d_bin,tuple;
    int n1,c1,h1,w1,r2,c2, i, j, k, l, n, r;
    float *a, *b, *c, *d;
  
    if (!enif_get_int(env, argv[0], &n1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &h1)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &w1)) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
    if (!enif_get_int(env, argv[5], &r2)) return enif_make_int(env,6);
    if (!enif_get_int(env, argv[6], &c2)) return enif_make_int(env,7);
    if (!enif_inspect_binary(env, argv[7], &b_bin )) return enif_make_int(env,8);
    if (!enif_get_int(env, argv[8], &n)) return enif_make_int(env,9);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n*c1*h1*w1 * sizeof(float), &c_bin);
    d = (float *) enif_make_new_binary(env, n*r2*c2 * sizeof(float), &d_bin);

    // random-select
    for(i=0;i<n;i++){
        r = rand() % n1;
        for(j=0;j<c1;j++){
            for(k=0;k<h1;k++){
                for(l=0;l<w1;l++){
                    c[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(r,j,k,l,c1,h1,w1)];
                }
            }
        }
        for(j=0;j<c2;j++){
            d[IDX2C(i,j,n)] = b[IDX2C(r,j,r2)];
        }    
    }

    tuple = enif_make_tuple2(env,c_bin,d_bin);
    return(tuple);
}

/*
random_select for 3D-tensor data
*/
static ERL_NIF_TERM
random_select3(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    ERL_NIF_TERM  c_bin,d_bin,tuple;
    int n1,h1,w1,r2,c2, i, j, k, n, r;
    float *a, *b, *c, *d;
  
    if (!enif_get_int(env, argv[0], &n1)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &h1)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &w1)) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[3], &a_bin )) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &r2)) return enif_make_int(env,5);
    if (!enif_get_int(env, argv[5], &c2)) return enif_make_int(env,6);
    if (!enif_inspect_binary(env, argv[6], &b_bin )) return enif_make_int(env,7);
    if (!enif_get_int(env, argv[7], &n)) return enif_make_int(env,8);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;
    c = (float *) enif_make_new_binary(env, n*h1*w1 * sizeof(float), &c_bin);
    d = (float *) enif_make_new_binary(env, n*r2*c2 * sizeof(float), &d_bin);

    // random-select
    for(i=0;i<n;i++){
        r = rand() % n1;
        for(j=0;j<h1;j++){
            for(k=0;k<w1;k++){
                c[IDX3C(i,j,k,h1,w1)] = a[IDX3C(r,j,k,h1,w1)];
            }
        }
        for(j=0;j<c2;j++){
            d[IDX2C(i,j,n)] = b[IDX2C(r,j,r2)];
        }    
    }

    tuple = enif_make_tuple2(env,c_bin,d_bin);
    return(tuple);
}



static ERL_NIF_TERM
is_near1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    int i, n, sw;
    float *a, *b;
  
    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;

    // near check
    sw = 0;
    for(i=0;i<n;i++){
       if(fabsf(a[i]) > fabsf(b[i])*1.15 || fabsf(a[i]) < fabsf(b[i])*0.85){
            printf("%f %f \r\n", a[i], b[i]);
            sw = 1;
        }
    }
    if(sw == 0)
        return enif_make_int(env,1); //true
    else
        return enif_make_int(env,0); //false
}

static ERL_NIF_TERM
is_equal1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin,b_bin;
    int i, n;
    float *a, *b;
  
    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);

    a = (float *) a_bin.data;
    b = (float *) b_bin.data;

    // equal check
    for(i=0;i<n;i++){
       if(a[i] != b[i]){
            return enif_make_int(env,0); //false
        }
    }
    
    return enif_make_int(env,1); //true
}



static ERL_NIF_TERM
analizer1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    int i, n, id;
    float *a;
    float max,min,sum;
  
    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &id)) return enif_make_int(env,3);

    a = (float *) a_bin.data;

    // near check
    for(i=0;i<n;i++){
        if(isnan(a[i])){
            return enif_make_int(env,9999);
        }
        if(isinf(a[i])){
            return enif_make_int(env,9998);
        }
    }

    //find max min avarage
    max = -999999999;
    min = 999999999;
    sum = 0;
    for(i=0;i<n;i++){
        if(a[i] > max)
            max = a[i];
        
        if(a[i] < min)
            min = a[i];
        
        sum = sum+a[i];

    }
    printf("id max min average\r\n");
    printf("%d %f %f %f \r\n", id, max, min, sum/(float)n);

    return enif_make_int(env,1);
}



  
  /*
  1st arg in_n of tensor
  2nd arg in_c of tensor
  3rd arg in_h of tensor
  4th arg in_w of tensor
  5th arg binary of tensor
  
  */
static ERL_NIF_TERM
standardize1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int in_n,in_c,in_h,in_w,n1,i,c1,h1,w1,count;
    float *a,*b;
    float sum,average;
  
    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
    if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);

    n1 = in_n * in_c * in_h * in_w;
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n1 * sizeof(float), &b_bin);
   
    
    for(i=0;i<in_n;i++){
        sum = 0.0;
        for(c1=0;c1<in_c;c1++){
            for(h1=0;h1<in_h;h1++){
                for(w1=0;w1<in_w;w1++){
                    sum = sum + a[IDX4C(i,c1,h1,w1,in_c,in_h,in_w)];
                }
            }
        }
        count = in_c * in_h * in_w;
        average = sum / (float)count;
        for(c1=0;c1<in_c;c1++){
            for(h1=0;h1<in_h;h1++){
                for(w1=0;w1<in_w;w1++){
                    b[IDX4C(i,c1,h1,w1,in_c,in_h,in_w)] = a[IDX4C(i,c1,h1,w1,in_c,in_h,in_w)] - average;
                }
            }
        }
    }
    
    return(b_bin);
}


  
/*
1st arg in_n of 3D tensor
2rd arg in_r of 3D tensor
3th arg in_c of 3D tensor
4th arg binary of tensor
5th arg nth in_r of 3D tensor
  
*/
static ERL_NIF_TERM
pickup1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int in_n,in_row,in_col,nth,n1,i,j;
    float *a,*b;
    
    if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_row)) return enif_make_int(env,2);
    if (!enif_get_int(env, argv[2], &in_col)) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[3], &a_bin )) return enif_make_int(env,4);
    if (!enif_get_int(env, argv[4], &nth)) return enif_make_int(env,5);

    n1 = in_n * in_col;
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n1 * sizeof(float), &b_bin);
     
      
    for(i=0;i<in_n;i++){
        for(j=0;j<in_col;j++){
            b[IDX2C(i,j,in_n)] = a[IDX3C(i,nth,j,in_row,in_col)];
        }
    }
      
    return(b_bin);
}
  

  
/*
1st arg size of tensor or matrix
2rd arg binary of tensor or matrix
  
*/
static ERL_NIF_TERM
copy1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin;
    int n,i;
    float *a,*b;
    
    if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
    if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);

    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
     
      
    for(i=0;i<n;i++){
        b[i] = a[i];
    }
      
    return(b_bin);
}


static ERL_NIF_TERM
slice1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  a_bin;
    ERL_NIF_TERM  b_bin,c_bin,d_bin,e_bin,tuple;
    int in_r,in_c,i,j,n,bias;
    float *a,*b,*c,*d,*e;
    
    if (!enif_get_int(env, argv[0], &in_r)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);

    n = in_r * (in_c / 4);
    a = (float *) a_bin.data;
    b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
    c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);
    d = (float *) enif_make_new_binary(env, n * sizeof(float), &d_bin);
    e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
     
      
    for(i=0;i<in_r;i++){
        for(j=0;j<n;j++){
            b[IDX2C(i,j,in_r)] = a[IDX2C(i,j,in_r)]; 
        }
    }
    bias = n;
    for(i=0;i<in_r;i++){
        for(j=0;j<n;j++){
            c[IDX2C(i,j,in_r)] = a[IDX2C(i,j+bias,in_r)]; 
        }
    }
    bias = 2*n;
    for(i=0;i<in_r;i++){
        for(j=0;j<n;j++){
            d[IDX2C(i,j,in_r)] = a[IDX2C(i,j+bias,in_r)]; 
        }
    }
    bias = 3*n;
    for(i=0;i<in_r;i++){
        for(j=0;j<n;j++){
            e[IDX2C(i,j,in_r)] = a[IDX2C(i,j+bias,in_r)]; 
        }
    }
      
    tuple = enif_make_tuple4(env,b_bin,c_bin,d_bin,e_bin);
    return(tuple);
}

static ERL_NIF_TERM
unslice1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
    ErlNifBinary  b_bin,c_bin,d_bin,e_bin;
    ERL_NIF_TERM  a_bin;
    int in_r,in_c,i,j,n,bias;
    float *a,*b,*c,*d,*e;
    
    if (!enif_get_int(env, argv[0], &in_r)) return enif_make_int(env,1);
    if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
    if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);
    if (!enif_inspect_binary(env, argv[3], &c_bin )) return enif_make_int(env,4);
    if (!enif_inspect_binary(env, argv[4], &d_bin )) return enif_make_int(env,5);
    if (!enif_inspect_binary(env, argv[5], &e_bin )) return enif_make_int(env,6);

    n = in_r * in_c * 4;
    a = (float *) enif_make_new_binary(env, n * sizeof(float), &a_bin);
    
      
    for(i=0;i<in_r;i++){
        for(j=0;j<n;j++){
            a[IDX2C(i,j,in_r)] = b[IDX2C(i,j,in_r)];
        }
    }
    bias = n;
    for(i=0;i<in_r;i++){
        for(j=0;j<n;j++){
            a[IDX2C(i,j+bias,in_r)] = c[IDX2C(i,j,in_r)]; 
        }
    }
    bias = 2*n;
    for(i=0;i<in_r;i++){
        for(j=0;j<n;j++){
            a[IDX2C(i,j+bias,in_r)] = d[IDX2C(i,j,in_r)] ; 
        }
    }
    bias = 3*n;
    for(i=0;i<in_r;i++){
        for(j=0;j<n;j++){
            a[IDX2C(i,j+bias,in_r)] = e[IDX2C(i,j,in_r)]; 
        }
    }
      
    return(a_bin);
}



// define the array of ErlNifFunc
static ErlNifFunc nif_funcs[] = {
  // {erl_function_name, erl_function_arity, c_function}
  {"mult1", 6, mult1},
  {"new1", 2, new1},
  {"new2", 3, new2},
  {"new3", 4, new3},
  {"new4", 5, new4},
  {"rand1", 1, rand1},
  {"add1", 3, add1},
  {"sub1", 3, sub1},
  {"emult1", 3, emult1},
  {"ediv1", 3, ediv1},
  {"transpose1", 3, transpose1},
  {"ident1", 1, ident1},
  {"activate_sigmoid", 2 ,activate_sigmoid},
  {"activate_tanh", 2 , activate_tanh},
  {"activate_relu", 2, activate_relu},
  {"activate_softmax", 3, activate_softmax},
  {"differ_sigmoid", 3, differ_sigmoid},
  {"differ_tanh", 3, differ_tanh},
  {"differ_relu", 3, differ_relu},
  {"smult1", 3, smult1},
  {"trace1", 3, trace1},
  {"mean_square", 4, mean_square},
  {"cross_entropy", 4, cross_entropy},
  {"elt1", 5, elt1},
  {"set1", 6, set1},
  {"add_diff1", 6, add_diff1},
  {"add_diff2", 10, add_diff2},
  {"average1", 3, average1},
  {"sum1", 3, sum1},
  {"to_list1", 3, to_list1},
  {"to_list2", 4, to_list2},
  {"to_list3", 5, to_list3},
  {"dropout1", 2 , dropout1},
  {"sgd1", 4, sgd1},
  {"momentum1", 5, momentum1},
  {"adagrad1", 5, adagrad1},
  {"rms1", 5, rms1},
  {"adam1", 6, adam1},
  {"accuracy1", 4, accuracy1},
  {"correct1", 4, correct1},
  {"pooling1", 7, pooling1},
  {"unpooling1", 8, unpooling1},
  {"convolute1", 13, convolute1},
  {"deconvolute1", 13, deconvolute1},
  {"deconvolute2", 13, deconvolute2},
  {"gradfilter1", 16, gradfilter1},
  {"gradfilter2", 16, gradfilter2},
  {"full1", 5, full1},
  {"unfull1", 5, unfull1},
  {"random_select1", 7, random_select1},
  {"random_select2", 9, random_select2},
  {"random_select3", 8, random_select3},
  {"is_near1", 3, is_near1},
  {"is_equal1", 3, is_equal1},
  {"analizer1", 3, analizer1},
  {"standardize1", 5, standardize1},
  {"pickup1", 5, pickup1},
  {"copy1", 2, copy1},
  {"slice1", 3, slice1},
  {"unslice1", 6, unslice1}
};

ERL_NIF_INIT(Elixir.Cumatrix, nif_funcs, NULL, NULL, NULL, NULL)

